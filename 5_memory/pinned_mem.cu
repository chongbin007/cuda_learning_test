#include "hip/hip_runtime.h"

#include "iostream"
#include <stdio.h>
 
using namespace std;

#define COPY_COUNTS 10
#define MEM_SIZE 25*1024*1024
 
float cuda_host_alloc_test(int size, bool up)
{
	//耗时统计
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
 
	int *a, *dev_a;
	//在主机上分配页锁定内存
    hipError_t cudaStatus = hipHostMalloc((void **)&a, size * sizeof(*a), hipHostMallocDefault);
	//在设备上分配内存空间
	cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(*dev_a));
	//计时开始
	hipEventRecord(start, 0);
 
	for (int i = 0; i < COPY_COUNTS; i++)
	{
		//从主机到设备复制数据
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		//从设备到主机复制数据
		cudaStatus = hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);

	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
 
	hipHostFree(a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);
 
	return (float)elapsedTime / 1000;
 
}
 
float cuda_host_Malloc_test(int size, bool up)
{
	//耗时统计
	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int *a, *dev_a;
 
	//在主机上分配可分页内存
	a = (int*)malloc(size * sizeof(*a));
 
	//在设备上分配内存空间
	hipError_t	cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(*dev_a));
 
	//计时开始
	hipEventRecord(start, 0);
 
	//执行从copy host to device 然后再 device to host执行100次，记录时间
	for (int i = 0; i < COPY_COUNTS; i++) {
		//从主机到设备复制数据
		cudaStatus = hipMemcpy(dev_a, a, size * sizeof(*dev_a), hipMemcpyHostToDevice);
		//从设备到主机复制数据
		cudaStatus = hipMemcpy(a, dev_a, size * sizeof(*dev_a), hipMemcpyDeviceToHost);

	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
 
	free(a);
	hipFree(dev_a);
	hipEventDestroy(start);
	hipEventDestroy(stop);
 
	return (float)elapsedTime / 1000;
}
 
int main()
{
	float allocTime = cuda_host_alloc_test(MEM_SIZE, true);
	cout << "页锁定内存: " << allocTime << " s" << endl;
	float mallocTime = cuda_host_Malloc_test(MEM_SIZE, true);
	cout << "可分页内存: " << mallocTime << " s" << endl;
	return 0;
}

// 反复拷贝数据进行性能测试
// 页锁定内存: 0.658992 s
// 可分页内存: 1.22233 s