#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define N (1024 * 1024)		  //每次从CPU传输到GPU的数据块大小
#define FULL_DATA_SIZE N * 20 //总数据量

__global__ void kernel(int *a, int *b, int *c)
{
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	//这里线程号应该小于FULL_DATA_SIZE
	if (threadID < FULL_DATA_SIZE)
	{
		c[threadID] = (a[threadID] + b[threadID]) / 2;
	}
}
//目的：计算两个数组，数组大小均为FULL_DATA_SIZE，的和
int main()
{

	int *host_a, *host_b, *host_c;
	int *dev_a, *dev_b, *dev_c;

	//在GPU上分配内存
	hipMalloc((void **)&dev_a, FULL_DATA_SIZE * sizeof(int));
	hipMalloc((void **)&dev_b, FULL_DATA_SIZE * sizeof(int));
	hipMalloc((void **)&dev_c, FULL_DATA_SIZE * sizeof(int));

	//在CPU上分配：可分页内存
	//数组大小FULL_DATA_SIZE
	host_a = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
	host_b = (int *)malloc(FULL_DATA_SIZE * sizeof(int));
	host_c = (int *)malloc(FULL_DATA_SIZE * sizeof(int));

	//主机上的两个数组随机赋值
	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = i;
		host_b[i] = FULL_DATA_SIZE - i;
	}

	// copy host to device
	hipMemcpy(dev_a, host_a, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, host_b, FULL_DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);
	std::cout << "启动 " << std::endl;

	hipDeviceSynchronize();
	//启动计时器
	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//启动函数，做数值加法
	kernel<<<FULL_DATA_SIZE / 1024, 1024>>>(dev_a, dev_b, dev_c);

	//数据拷贝回主机
	hipMemcpy(host_c, dev_c, FULL_DATA_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	std::cout << "event计时： " << elapsedTime << "ms" << std::endl;

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}
//event计时： 111.983ms